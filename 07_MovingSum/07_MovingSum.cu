#include "hip/hip_runtime.h"
/*
* This program will calculate a moving sum over an vector, where each output is
* the accumulation of a sliding window. In other words, the input for each
* output is defined by the same global index but also accumulates the items left
* and right of it of the size RADIUS.
*
*    ┌─────────────────────────────┐
*    │1 1 1 1 1 1 1 1 1 1 1 1 1 1 1│
*    └▲─▲─────▲─▲─▲───▲─▲─▲────────┘
*     │ │     │ │ │   │ │ │
*     ├─┘     └─┼─┘   └─┼─┘
*     │         │       │-->RADIUS=1
*     │         │       │
*     │         │       │
*    ┌┴─────────┴───────┴───────────┐
*    │2 3 3 3 3 3 3 3 3 3 3 3 3 3 2 │
*    └──────────────────────────────┘
*    * Only three sliding window are drawn here, not all of them
*
*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>

using namespace std;

#define RADIUS      10
#define BLOCKSIZE   512
#define WIDTH       65536

/* This is our CUDA call wrapper, we will use in PAC.
*
*  Almost all CUDA calls should be wrapped with this makro.
*  Errors from these calls will be catched and printed on the console.
*  If an error appears, the program will terminate.
*
* Example: gpuErrCheck(hipMalloc(&deviceA, N * sizeof(int)));
*          gpuErrCheck(hipMemcpy(deviceA, hostA, N * sizeof(int), hipMemcpyHostToDevice));
*/
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort)
        {
            exit(code);
        }
    }
}


/*
*
* Rewrite the kernel movingSumGlobal using a static allocation of shared
* memory, as shown in the slides of the lecture.
*
*/
__global__ void movingSumSharedMemStatic(int* vec, int* result_vec, int size) //size = 10.. so sum includes i-10.....i....i+10
{
    // Assuming BLOCKSIZE and RADIUS are defined appropriately.
    __shared__ int shm_vec[BLOCKSIZE + 2 * RADIUS];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local_index = threadIdx.x + RADIUS; // Position in shared memory

    // Load the main data element for the current thread
    if(index < size) {
        shm_vec[local_index] = vec[index];

        if(threadIdx.x < RADIUS) { //use the n-threads (RADIUIS ones) to handle left and right borders
            // left halo handling
            if( index < RADIUS ) { // thread all to the left (there are no left neighbours)
                shm_vec[threadIdx.x] = 0;
            }
            else {
                shm_vec[threadIdx.x] = vec[index - RADIUS];
            }

            //right halo handling
            if (index + BLOCKSIZE >= size) { //if last block there are no neighbours
                shm_vec[BLOCKSIZE + local_index] = 0;
            }
            else {
                shm_vec[BLOCKSIZE + local_index] = vec[index + BLOCKSIZE];
            }
        }
    }
    
    __syncthreads(); // Ensure all threads have loaded data into shared memory

    // Compute the moving sum using the shared memory indices
    int tmp = 0;
    for (int i = local_index - RADIUS; i <= local_index + RADIUS; i++) {
        tmp += shm_vec[i];
    }
    
    // Store the result, ensuring we don't write out-of-bounds
    if(index < size)
        result_vec[index] = tmp;
}


/*
*
* Rewrite the kernel movingSumGlobal using a dynamic allocation of shared
* memory, more information can be found here:
* https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared
*
* You should use something like this:
*    extern __shared__ int shmVec[];
* and extend the execution configuration with the size of the shmVec.
*
<<< n_blocks, n_threads, size shm >>>
annahme:
- threads 1024
- shm muss links und rechts +10 elemente haben 
*/
/**
 * Threads that belong to a thread block cluster, can read, write or perform atomics in the distributed address space, regardless whether the address belongs to the local thread block or a remote thread block. 
 * Whether a kernel uses distributed shared memory or not, the shared memory size specifications, static or dynamic is still per thread block. The size of distributed shared memory is just the number of thread blocks per cluster multiplied by the size of shared memory per thread block. 
 **/
__global__ void movingSumSharedMemDynamic(int* vec, int* result_vec, int size)
{
    extern __shared__ int shm_vec[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int local_index = threadIdx.x + RADIUS; // Position in shared memory

    // Load the main data element for the current thread
    if(index < size) {
        shm_vec[local_index] = vec[index];

        if(threadIdx.x < RADIUS) { //use the n-threads (RADIUIS ones) to handle left and right borders
            // left halo handling
            if( index < RADIUS ) { // thread all to the left (there are no left neighbours)
                shm_vec[threadIdx.x] = 0;
            }
            else {
                shm_vec[threadIdx.x] = vec[index - RADIUS];
            }

            //right halo handling
            if (index + BLOCKSIZE >= size) { //if last block there are no neighbours
                shm_vec[BLOCKSIZE + local_index] = 0;
            }
            else {
                shm_vec[BLOCKSIZE + local_index] = vec[index + BLOCKSIZE];
            }
        }
    }
    
    __syncthreads(); // Ensure all threads have loaded data into shared memory

    // Compute the moving sum using the shared memory indices
    int tmp = 0;
    for (int i = local_index - RADIUS; i <= local_index + RADIUS; i++) {
        tmp += shm_vec[i];
    }
    
    // Store the result, ensuring we don't write out-of-bounds
    if(index < size)
        result_vec[index] = tmp;
}


/*
*
* Rewrite the kernel movingSumGlobal using only global memory and no
* shared mem. Use atomic add operations.
* https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
*
* You must reverse the access pattern, such that without atomics, conflicts can occure.
* So 1 Thread writes "its" value into multiple outputs.
*/
__global__ void movingSumAtomics(int* vec, int* result_vec, int size)
{
    int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int offset = -RADIUS; offset <= RADIUS; offset++) { //a thread writes to all its neighbours his additional value
        int internalIdx = globalIdx + offset;
        if (internalIdx >= 0 && internalIdx < size) {
            atomicAdd(result_vec + internalIdx, vec[globalIdx]);
        }
    }
}


// This is the GPU refernece implementation
__global__ void movingSumGlobal(int* vec, int* result_vec, int size)
{

    int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int result = 0;
    if (globalIdx >= RADIUS && globalIdx < size - RADIUS) {
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            result += vec[globalIdx + offset];
        }
    }

    if (globalIdx < RADIUS) {
        for (int offset = 0 - globalIdx; offset <= RADIUS; offset++) {
            result += vec[globalIdx + offset];
        }
    }

    if (globalIdx < size && globalIdx >= size - RADIUS) {
        for (int offset = -RADIUS; offset < size - globalIdx; offset++) {
            result += vec[globalIdx + offset];
        }
    }

    result_vec[globalIdx] = result;
}


// CPU reference implementation
void movingSumCPU(int* vec, int* result_vec, int size)
{
    int result;

    for (int i = 0; i < size; ++i) {
        result = 0;

        if (i >= RADIUS && i < size - RADIUS) {
            for (int offset = -RADIUS; offset <= RADIUS; offset++) {
                result += vec[i + offset];
            }
        }

        if (i < RADIUS) {
            for (int offset = 0 - i; offset <= RADIUS; offset++) {
                result += vec[i + offset];
            }
        }

        if (i < size && i >= size - RADIUS) {
            for (int offset = -RADIUS; offset < size - i; offset++) {
                result += vec[i + offset];
            }
        }

        result_vec[i] = result;
    }
}


// Compare result vectors
int compareResultVec(int* vectorCPU, int* vectorGPU, int size)
{
    int error = 0;
    for (int i = 0; i < size; i++)
    {
        error += abs(vectorCPU[i] - vectorGPU[i]);
    }
    if (error == 0)
    {
        cout << "No errors. All good!" << endl;
        return 0;
    }
    else
    {
        cout << "Accumulated error: " << error << endl;
        return -1;
    }
}


int main(void)
{
    // Allocate and prepare input vector on host memory
    int* hostVecInput = new int[WIDTH];
    int* hostVecOutputCPU = new int[WIDTH];
    int* hostVecOutputGPU1 = new int[WIDTH];
    int* hostVecOutputGPU2 = new int[WIDTH];
    int* hostVecOutputGPU3 = new int[WIDTH];
    int* hostVecOutputGPU4 = new int[WIDTH];

    for (int i = 0; i < WIDTH; i++) {
        hostVecInput[i] = 1;
    }

    // Get the CPU result
    movingSumCPU(hostVecInput, hostVecOutputCPU, WIDTH);

    // Allocate device memory
    int* deviceVecInput;
    int* deviceVecOutput1;
    int* deviceVecOutput2;
    int* deviceVecOutput3;
    int* deviceVecOutput4;
    gpuErrCheck(hipMalloc(&deviceVecInput, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput1, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput2, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput3, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput4, WIDTH * sizeof(int)));

    // Copy data from host to device
    gpuErrCheck(hipMemcpy(deviceVecInput, hostVecInput, WIDTH * sizeof(int), hipMemcpyHostToDevice));

    // Run kernel on all elements on the GPU
    int nbr_blocks = ((WIDTH % BLOCKSIZE) != 0) ? (WIDTH / BLOCKSIZE + 1) : (WIDTH / BLOCKSIZE);
    //WIDTH = array-elems
    movingSumGlobal << <nbr_blocks, BLOCKSIZE >> > (deviceVecInput, deviceVecOutput1, WIDTH);
    gpuErrCheck(hipPeekAtLastError());
    movingSumSharedMemStatic << <nbr_blocks, BLOCKSIZE >> > (deviceVecInput, deviceVecOutput2, WIDTH);
    gpuErrCheck(hipPeekAtLastError());
    //configuration how much shared memory is needed is passed in the kernel call << nbr_blocks, BLOCKSIZE, size shmVec >
    size_t shmSize = (BLOCKSIZE + 2 * RADIUS) * sizeof(int);
    movingSumSharedMemDynamic <<<nbr_blocks, BLOCKSIZE, shmSize >>> (deviceVecInput, deviceVecOutput3, WIDTH);
    gpuErrCheck(hipPeekAtLastError());
    movingSumAtomics << <nbr_blocks, BLOCKSIZE >> > (deviceVecInput, deviceVecOutput4, WIDTH);
    gpuErrCheck(hipPeekAtLastError());

    // Copy the result stored in device_y back to host_y
    gpuErrCheck(hipMemcpy(hostVecOutputGPU1, deviceVecOutput1, WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrCheck(hipMemcpy(hostVecOutputGPU2, deviceVecOutput2, WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrCheck(hipMemcpy(hostVecOutputGPU3, deviceVecOutput3, WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrCheck(hipMemcpy(hostVecOutputGPU4, deviceVecOutput4, WIDTH * sizeof(int), hipMemcpyDeviceToHost));

    // Check for errors in result
    auto ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU1, WIDTH);
    ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU2, WIDTH);
    ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU3, WIDTH);
    ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU4, WIDTH);

    // Free memory on device & host
    hipFree(deviceVecInput);
    hipFree(deviceVecOutput1);
    hipFree(deviceVecOutput2);
    hipFree(deviceVecOutput3);
    hipFree(deviceVecOutput4);
    delete[] hostVecInput;
    delete[] hostVecOutputCPU;
    delete[] hostVecOutputGPU1;
    delete[] hostVecOutputGPU2;
    delete[] hostVecOutputGPU3;
    delete[] hostVecOutputGPU4;

    return 0;
}