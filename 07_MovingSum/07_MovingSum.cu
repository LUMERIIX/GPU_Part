#include "hip/hip_runtime.h"
/*
* This program will calculate a moving sum over an vector, where each output is
* the accumulation of a sliding window. In other words, the input for each
* output is defined by the same global index but also accumulates the items left
* and right of it of the size RADIUS.
*
*    ┌─────────────────────────────┐
*    │1 1 1 1 1 1 1 1 1 1 1 1 1 1 1│
*    └▲─▲─────▲─▲─▲───▲─▲─▲────────┘
*     │ │     │ │ │   │ │ │
*     ├─┘     └─┼─┘   └─┼─┘
*     │         │       │-->RADIUS=1
*     │         │       │
*     │         │       │
*    ┌┴─────────┴───────┴───────────┐
*    │2 3 3 3 3 3 3 3 3 3 3 3 3 3 2 │
*    └──────────────────────────────┘
*    * Only three sliding window are drawn here, not all of them
*
*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>

using namespace std;

#define RADIUS      10
#define BLOCKSIZE   512
#define WIDTH       65536

/* This is our CUDA call wrapper, we will use in PAC.
*
*  Almost all CUDA calls should be wrapped with this makro.
*  Errors from these calls will be catched and printed on the console.
*  If an error appears, the program will terminate.
*
* Example: gpuErrCheck(hipMalloc(&deviceA, N * sizeof(int)));
*          gpuErrCheck(hipMemcpy(deviceA, hostA, N * sizeof(int), hipMemcpyHostToDevice));
*/
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort)
        {
            exit(code);
        }
    }
}


/*
*
* Rewrite the kernel movingSumGlobal using a static allocation of shared
* memory, as shown in the slides of the lecture.
*
*/
__global__ void movingSumSharedMemStatic(int* vec, int* result_vec, int size) //size = 10.. so sum includes i-10.....i....i+10
{
    //block has 512Threads
    __shared__ int shm_vec[BLOCKSIZE + 2*RADIUS]; //add 2x radius for the left and right border

    int index = blockIdx.x * blockDim.x + threadIdx.x; //standard line
    shm_vec[threadIdx.x + RADIUS] = vec[index]; //treat 

    if(blockIdx.x != 0 && blockIdx.x != ( WIDTH / BLOCKSIZE ) -1 ) { // if not a border-block load angrenzende elemente (for block idx 1 to n-1)
        if(threadIdx.x < 10 ) { // (left) only execute on threads which need edge blocks!
            for(int i = 0; i < RADIUS; i++) {
                shm_vec[i] = result_vec[index-RADIUS+i];
            }
        }
        else if(threadIdx.x > BLOCKSIZE-RADIUS) {
            for(int i = 0; i < RADIUS; i++) {
                shm_vec[BLOCKSIZE+i] = result_vec[index+RADIUS+i];
            }
        }
    }
    else if(blockIdx.x == 0) { // if first block load angrenzende elemente (for block idx 1 to n-1)
        for(int i = 0; i < RADIUS; i++) {
            shm_vec[i] = 0;
        }
    }
    else if(blockIdx.x == ( WIDTH / BLOCKSIZE ) -1) { // if last block load angrenzende elemente (for block idx 1 to n-1)
        for(int i = 0; i < RADIUS; i++) {
            shm_vec[BLOCKSIZE+i] = 0;
        }
    }
    shm_vec[threadIdx.x + RADIUS] = vec[index]; //treat the current element
    __syncthreads(); //wait for all threads to load their data

    int tmp = 0;
    for(int i = index-RADIUS; i < index+RADIUS+1; i++) { //sum the elements
        tmp += shm_vec[i];
    }

    result_vec[index] = tmp; //write the result


    //ToDo
    //1) create shared cudaMemory
    //2) load data from vec to shm_vec
    //3)!!!attention broder left and right
    //int tmp = 0;
    //for i in readiu;
    //    tmp += shm_vec[i]
//
    //result_vec[i] = tmp
}


/*
*
* Rewrite the kernel movingSumGlobal using a dynamic allocation of shared
* memory, more information can be found here:
* https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared
*
* You should use something like this:
*    extern __shared__ int shmVec[];
* and extend the execution configuration with the size of the shmVec.
*
<<< n_blocks, n_threads, size shm >>>
annahme:
- threads 1024
- shm muss links und rechts +10 elemente haben 
*/
__global__ void movingSumSharedMemDynamic(int* vec, int* result_vec, int size)
{
    //ToDo
}


/*
*
* Rewrite the kernel movingSumGlobal using only global memory and no
* shared mem. Use atomic add operations.
* https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
*
* You must reverse the access pattern, such that without atomics, conflicts can occure.
* So 1 Thread writes "its" value into multiple outputs.
*/
__global__ void movingSumAtomics(int* vec, int* result_vec, int size)
{
    //ToDo
}


// This is the GPU refernece implementation
__global__ void movingSumGlobal(int* vec, int* result_vec, int size)
{

    int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int result = 0;
    if (globalIdx >= RADIUS && globalIdx < size - RADIUS) {
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            result += vec[globalIdx + offset];
        }
    }

    if (globalIdx < RADIUS) {
        for (int offset = 0 - globalIdx; offset <= RADIUS; offset++) {
            result += vec[globalIdx + offset];
        }
    }

    if (globalIdx < size && globalIdx >= size - RADIUS) {
        for (int offset = -RADIUS; offset < size - globalIdx; offset++) {
            result += vec[globalIdx + offset];
        }
    }

    result_vec[globalIdx] = result;
}


// CPU reference implementation
void movingSumCPU(int* vec, int* result_vec, int size)
{
    int result;

    for (int i = 0; i < size; ++i) {
        result = 0;

        if (i >= RADIUS && i < size - RADIUS) {
            for (int offset = -RADIUS; offset <= RADIUS; offset++) {
                result += vec[i + offset];
            }
        }

        if (i < RADIUS) {
            for (int offset = 0 - i; offset <= RADIUS; offset++) {
                result += vec[i + offset];
            }
        }

        if (i < size && i >= size - RADIUS) {
            for (int offset = -RADIUS; offset < size - i; offset++) {
                result += vec[i + offset];
            }
        }

        result_vec[i] = result;
    }
}


// Compare result vectors
int compareResultVec(int* vectorCPU, int* vectorGPU, int size)
{
    int error = 0;
    for (int i = 0; i < size; i++)
    {
        error += abs(vectorCPU[i] - vectorGPU[i]);
    }
    if (error == 0)
    {
        cout << "No errors. All good!" << endl;
        return 0;
    }
    else
    {
        cout << "Accumulated error: " << error << endl;
        return -1;
    }
}


int main(void)
{
    // Allocate and prepare input vector on host memory
    int* hostVecInput = new int[WIDTH];
    int* hostVecOutputCPU = new int[WIDTH];
    int* hostVecOutputGPU1 = new int[WIDTH];
    int* hostVecOutputGPU2 = new int[WIDTH];
    int* hostVecOutputGPU3 = new int[WIDTH];
    int* hostVecOutputGPU4 = new int[WIDTH];

    for (int i = 0; i < WIDTH; i++) {
        hostVecInput[i] = 1;
    }

    // Get the CPU result
    movingSumCPU(hostVecInput, hostVecOutputCPU, WIDTH);

    // Allocate device memory
    int* deviceVecInput;
    int* deviceVecOutput1;
    int* deviceVecOutput2;
    int* deviceVecOutput3;
    int* deviceVecOutput4;
    gpuErrCheck(hipMalloc(&deviceVecInput, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput1, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput2, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput3, WIDTH * sizeof(int)));
    gpuErrCheck(hipMalloc(&deviceVecOutput4, WIDTH * sizeof(int)));

    // Copy data from host to device
    gpuErrCheck(hipMemcpy(deviceVecInput, hostVecInput, WIDTH * sizeof(int), hipMemcpyHostToDevice));

    // Run kernel on all elements on the GPU
    int nbr_blocks = ((WIDTH % BLOCKSIZE) != 0) ? (WIDTH / BLOCKSIZE + 1) : (WIDTH / BLOCKSIZE);
    //WIDTH = array-elems
    movingSumGlobal << <nbr_blocks, BLOCKSIZE >> > (deviceVecInput, deviceVecOutput1, WIDTH);
    gpuErrCheck(hipPeekAtLastError());
    movingSumSharedMemStatic << <nbr_blocks, BLOCKSIZE >> > (deviceVecInput, deviceVecOutput2, WIDTH);
    gpuErrCheck(hipPeekAtLastError());
    //ToDo: movingSumSharedMemDynamic <<<nbr_blocks, BLOCKSIZE, ?????????? >>> (deviceVecInput, deviceVecOutput3, WIDTH);
    gpuErrCheck(hipPeekAtLastError());
    //ToDo: movingSumAtomics << <nbr_blocks, BLOCKSIZE >> > (deviceVecInput, deviceVecOutput4, WIDTH);
    gpuErrCheck(hipPeekAtLastError());

    // Copy the result stored in device_y back to host_y
    gpuErrCheck(hipMemcpy(hostVecOutputGPU1, deviceVecOutput1, WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrCheck(hipMemcpy(hostVecOutputGPU2, deviceVecOutput2, WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrCheck(hipMemcpy(hostVecOutputGPU3, deviceVecOutput3, WIDTH * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrCheck(hipMemcpy(hostVecOutputGPU4, deviceVecOutput4, WIDTH * sizeof(int), hipMemcpyDeviceToHost));

    // Check for errors in result
    auto ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU1, WIDTH);
    ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU2, WIDTH);
    ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU3, WIDTH);
    ret = compareResultVec(hostVecOutputCPU, hostVecOutputGPU4, WIDTH);

    // Free memory on device & host
    hipFree(deviceVecInput);
    hipFree(deviceVecOutput1);
    hipFree(deviceVecOutput2);
    hipFree(deviceVecOutput3);
    hipFree(deviceVecOutput4);
    delete[] hostVecInput;
    delete[] hostVecOutputCPU;
    delete[] hostVecOutputGPU1;
    delete[] hostVecOutputGPU2;
    delete[] hostVecOutputGPU3;
    delete[] hostVecOutputGPU4;

    return 0;
}